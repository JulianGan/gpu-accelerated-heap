#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <assert.h>

// array implementation of a max heap
struct PriorityQueue{
	float * data;
	size_t size;
	size_t max_size;
};

/* FUNCTION DECLARATION */
struct PriorityQueue * new_PriorityQueue(size_t max_size);
void delete_PriorityQueue(struct PriorityQueue * queue);
float peek(struct PriorityQueue * queue);
bool push(struct PriorityQueue * queue, float item);
float pop(struct PriorityQueue * queue);

/* UTILITY FUNCTION */
void swap(float * a, float * b){
	float temp;
	temp = * a;
	* a = * b;
	* b = temp;
}

void print_PriorityQueue(struct PriorityQueue * queue){
	for (int i = 0; i < queue->size; i++){
		printf("%f ", queue->data[i]);
	}
	printf("\n");
}

int main(int argc, char *argv[]){

	struct PriorityQueue * queue = new_PriorityQueue(10);
	assert(queue != NULL);

	int i;
	float j;
	for (i = 0; i < 10; i++){
		push(queue, (float) i);
	}
	for (i = 0; i < 10; i++){
		j = pop(queue);
		if (!isnan(j)) printf("%f\n", j);
	}

	delete_PriorityQueue(queue);

	return 0;
}

struct PriorityQueue * new_PriorityQueue(size_t max_size){

	struct PriorityQueue * queue;
	queue = (struct PriorityQueue *) malloc(sizeof(struct PriorityQueue));
	if (queue == NULL) return NULL;

	queue->data = (float *) malloc(max_size * sizeof(float));
	if (queue->data == NULL) return NULL;

	queue->size = 0;
	queue->max_size = max_size;

	return queue;
}

void delete_PriorityQueue(struct PriorityQueue * queue){
	if (queue != NULL){
		free(queue->data);
		free(queue);
	}
}

float peek(struct PriorityQueue * queue){
	/*
	returns the first item in the queue
	if there are no items return NaN
	*/
	if (queue->size > 0) return queue->data[0];
	return NAN; 
}

bool push(struct PriorityQueue * queue, float item){
	/*
	inserts an item into the queue
	returns 1 on success
	returns 0 on error
	*/

	// check capacity
	if (queue->size == queue->max_size) return false;

	// insert item
	queue->data[queue->size] = item;
	++queue->size;

	// swim up
	size_t idx = queue->size - 1;
	while (idx > 0 && queue->data[idx] > queue->data[(idx - 1) / 2]){
		swap(&queue->data[idx], &queue->data[(idx - 1) / 2]);
		idx -= 1; idx /= 2;
	}

	return true;
}

float pop(struct PriorityQueue * queue){
	/*
	pop first item in queue and return its value
	if there are no items return NaN
	*/

	// check if there are any items to pop
	if (queue->size == 0) return NAN;
	
	// get item
	float ans = queue->data[0];

	// pop item
	swap(&queue->data[0], &queue->data[--queue->size]);

	// swim down
	size_t n = queue->size;
	size_t idx = 0;
	size_t l, r;
	while (true){
		l = idx * 2 + 1;
		r = idx * 2 + 2;
		if (l < n && queue->data[idx] < queue->data[l] && r < n && queue->data[idx] < queue->data[r]){
			if (queue->data[l] > queue->data[r]){
				swap(&queue->data[idx], &queue->data[l]);
				idx = l;
			} else {
				swap(&queue->data[idx], &queue->data[r]);
				idx = r;
			}
			continue;
		}
		if (l < n && queue->data[idx] < queue->data[l]){
			swap(&queue->data[idx], &queue->data[l]);
			idx = l;
			continue;
		}
		if (r < n && queue->data[idx] < queue->data[r]){
			swap(&queue->data[idx], &queue->data[r]);
			idx = r;
			continue;
		}
		break;
	}

	return ans;
}